#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <float.h>
#include <hiprand/hiprand_kernel.h>
#include "rt/vector.cuh"
#include "rt/ray.cuh"
#include "rt/sphere.cuh"
#include "rt/hittable_list.cuh"
#include "rt/camera.cuh"
#include "rt/material.cuh"


#define CUDA_CALL(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

// Matching the C++ code would recurse enough into rayColor() calls that
// it was blowing up the stack, so we have to turn this into a
// limited-depth loop instead.  Later code in the book limits to a max
// depth of 50, so we adapt this a few chapters early on the GPU.
__device__ vector rayColor(const ray& r, hittable **world, hiprandState *local_rand_state) {
    ray cur_ray = r;
    vector cur_attenuation = vector(1.0,1.0,1.0);
    for(int i = 0; i < 50; i++) {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            vector attenuation;
            if(rec.matPtr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else {
                return vector(0.0,0.0,0.0);
            }
        }
        else {
            vector unit_direction = unitVector(cur_ray.direction());
            float t = 0.5f*(unit_direction.y() + 1.0f);
            vector c = (1.0f-t)*vector(1.0, 1.0, 1.0) + t*vector(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }
    }
    return vector(0.0,0.0,0.0); // exceeded recursion
}

__global__ void rand_init(hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    // Original: Each thread gets same seed, a different sequence number, no offset
    // hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
    // BUGFIX, see Issue#2: Each thread gets different seed, same sequence for
    // performance improvement of about 2x!
    hiprand_init(1984+pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void render(vector *fb, int max_x, int max_y, int ns, camera **cam, hittable **world, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    vector col(0,0,0);
    for(int s=0; s < ns; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->getRay(u, v, &local_rand_state);
        col += rayColor(r, world, &local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    fb[pixel_index] = col;
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_world(hittable **d_list, hittable **d_world, camera **d_camera, int nx, int ny, hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;
        d_list[0] = new sphere(vector(0,-1000.0,-1), 1000,
                               new lambertian(vector(0.5, 0.5, 0.5)));
        int i = 1;
        for(int a = -11; a < 11; a++) {
            for(int b = -11; b < 11; b++) {
                float choose_mat = RND;
                vector center(a+RND,0.2,b+RND);
                if(choose_mat < 0.8f) {
                    d_list[i++] = new sphere(center, 0.2,
                                             new lambertian(vector(RND*RND, RND*RND, RND*RND)));
                }
                else if(choose_mat < 0.95f) {
                    d_list[i++] = new sphere(center, 0.2,
                                             new metal(vector(0.5f*(1.0f+RND), 0.5f*(1.0f+RND), 0.5f*(1.0f+RND)), 0.5f*RND));
                }
                else {
                    d_list[i++] = new sphere(center, 0.2, new dielectric(1.5));
                }
            }
        }
        d_list[i++] = new sphere(vector(0, 1,0),  1.0, new dielectric(1.5));
        d_list[i++] = new sphere(vector(-4, 1, 0), 1.0, new lambertian(vector(0.4, 0.2, 0.1)));
        d_list[i++] = new sphere(vector(4, 1, 0),  1.0, new metal(vector(0.7, 0.6, 0.5), 0.0));
        *rand_state = local_rand_state;
        *d_world  = new hittable_list(d_list, 22*22+1+3);

        vector lookfrom(13,2,3);
        vector lookat(0,0,0);
        float dist_to_focus = 10.0; (lookfrom-lookat).length();
        float aperture = 0.1;
        *d_camera   = new camera(lookfrom,
                                 lookat,
                                 vector(0,1,0),
                                 30.0,
                                 float(nx)/float(ny),
                                 aperture,
                                 dist_to_focus);
    }
}

__global__ void free_world(hittable **d_list, hittable **d_world, camera **d_camera) {
    for(int i=0; i < 22*22+1+3; i++) {
        delete ((sphere *)d_list[i])->matPtr;
        delete d_list[i];
    }
    delete *d_world;
    delete *d_camera;
}

int main() {
    int nx = 1200;
    int ny = 800;
    int ns = 10;
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << nx << "x" << ny << " image with " << ns << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx*ny;
    size_t fb_size = num_pixels*sizeof(vector);

    // allocate FB
    vector *fb;
    CUDA_CALL(hipMallocManaged((void **)&fb, fb_size));

    // allocate random state
    hiprandState *d_rand_state;
    CUDA_CALL(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));
    hiprandState *d_rand_state2;
    CUDA_CALL(hipMalloc((void **)&d_rand_state2, 1*sizeof(hiprandState)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init<<<1,1>>>(d_rand_state2);
    CUDA_CALL(hipGetLastError());
    CUDA_CALL(hipDeviceSynchronize());

    // make our world of hittables & the camera
    hittable **d_list;
    int num_hittables = 22*22+1+3;
    CUDA_CALL(hipMalloc((void **)&d_list, num_hittables*sizeof(hittable *)));
    hittable **d_world;
    CUDA_CALL(hipMalloc((void **)&d_world, sizeof(hittable *)));
    camera **d_camera;
    CUDA_CALL(hipMalloc((void **)&d_camera, sizeof(camera *)));
    create_world<<<1,1>>>(d_list, d_world, d_camera, nx, ny, d_rand_state2);
    CUDA_CALL(hipGetLastError());
    CUDA_CALL(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);
    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);
    CUDA_CALL(hipGetLastError());
    CUDA_CALL(hipDeviceSynchronize());
    render<<<blocks, threads>>>(fb, nx, ny,  ns, d_camera, d_world, d_rand_state);
    CUDA_CALL(hipGetLastError());
    CUDA_CALL(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output FB as Image
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*nx + i;
            int ir = int(255.99*fb[pixel_index].r());
            int ig = int(255.99*fb[pixel_index].g());
            int ib = int(255.99*fb[pixel_index].b());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    // clean up
    CUDA_CALL(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list,d_world,d_camera);
    CUDA_CALL(hipGetLastError());
    CUDA_CALL(hipFree(d_camera));
    CUDA_CALL(hipFree(d_world));
    CUDA_CALL(hipFree(d_list));
    CUDA_CALL(hipFree(d_rand_state));
    CUDA_CALL(hipFree(d_rand_state2));
    CUDA_CALL(hipFree(fb));

    hipDeviceReset();
}