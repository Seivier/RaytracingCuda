#include "hip/hip_runtime.h"
//
// Created by vigb9 on 20/07/2023.
//

#include "Raytracing/Material.cuh"
#include "Raytracing/Hittable.cuh"

bool Lambertian::scatter(const Ray& r_in, const HitRecord& rec, Color& attenuation, Ray& scattered) const
{
	auto scatter_direction = rec.normal + randomUnitVector();
	// Catch degenerate scatter direction
	if (scatter_direction.nearZero())
		scatter_direction = rec.normal;
	scattered = Ray(rec.p, scatter_direction);
	attenuation = albedo;
	return true;
}

bool Metal::scatter(const Ray& r_in, const HitRecord& rec, Color& attenuation, Ray& scattered) const
{
	Vector reflected = reflect(unitVector(r_in.direction()), rec.normal);
	scattered = Ray(rec.p, reflected + fuzz*randomInUnitSphere());
	attenuation = albedo;
	return (dot(scattered.direction(), rec.normal) > 0);
}

bool Dielectric::scatter(const Ray& r_in, const HitRecord& rec, Color& attenuation, Ray& scattered) const
{
	attenuation = Color(1.0f, 1.0f, 1.0f);
	float refraction_ratio = rec.frontFace ? (1.0f / ir) : ir;

	Vector unit_direction = unitVector(r_in.direction());
	float cos_theta = fmin(dot(-unit_direction, rec.normal), 1.0f);
	float sin_theta = sqrt(1.0f - cos_theta*cos_theta);

	bool cannot_refract = refraction_ratio * sin_theta > 1.0f;
	Vector direction;

	if (cannot_refract || reflectance(cos_theta, refraction_ratio) > randomFloat())
		direction = reflect(unit_direction, rec.normal);
	else
		direction = refract(unit_direction, rec.normal, refraction_ratio);

	scattered = Ray(rec.p, direction);
	return true;
}

float Dielectric::reflectance(float cosine, float ref_idx)
{
	// Use Schlick's approximation for reflectance.
	auto r0 = (1 - ref_idx) / (1 + ref_idx);
	r0 = r0*r0;
	return r0 + (1 - r0)*pow((1 - cosine), 5);
}
